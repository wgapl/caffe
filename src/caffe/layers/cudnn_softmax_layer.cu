
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
<<<<<<< HEAD
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
=======
	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	Dtype alpha = 1.0;
	Dtype beta = 0.0;
	CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
			CUDNN_SOFTMAX_MODE_CHANNEL,
			reinterpret_cast<void *>(&alpha),
			bottom_desc_, bottom_data,
			reinterpret_cast<void *>(&beta),
			top_desc_, top_data));
>>>>>>> lrcn/recurrent
}

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

<<<<<<< HEAD
    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
          CUDNN_SOFTMAX_MODE_CHANNEL,
          cudnn::dataType<Dtype>::one,
          top_desc_, top_data, top_desc_, top_diff,
          cudnn::dataType<Dtype>::zero,
=======
    Dtype alpha = 1.0;
    Dtype beta = 0.0;

//    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
//        CUDNN_SOFTMAX_MODE_CHANNEL,
//        top_desc_, top_data, top_desc_, top_diff, bottom_desc_, bottom_diff));

    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
          CUDNN_SOFTMAX_MODE_CHANNEL,
          reinterpret_cast<void *>(&alpha),
          top_desc_, top_data, top_desc_, top_diff,
          reinterpret_cast<void *>(&beta),
>>>>>>> lrcn/recurrent
          bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
